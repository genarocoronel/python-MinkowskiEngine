#include "hip/hip_runtime.h"
#include "gpu.cuh"
#include "pruning.cuh"

template <typename Dtype, typename Itype>
__global__ void copy_in_out_map(const int n, const Dtype *in_feat,
                                Dtype *out_feat, int nchannel,
                                const Itype *in_map, const Itype *out_map) {
  CUDA_KERNEL_LOOP(index, n) {
    int nrow = index / nchannel;
    int ch = index % nchannel;
    out_feat[out_map[nrow] * nchannel + ch] =
        in_feat[in_map[nrow] * nchannel + ch];
  }
}

template <typename Dtype, typename Itype>
void PruningForwardKernelGPU(const Dtype *d_in_feat, Dtype *d_out_feat,
                             int nchannel,
                             const std::vector<std::vector<Itype>> &in_maps,
                             const std::vector<std::vector<Itype>> &out_maps,
                             hipStream_t stream) {
  int nnz = in_maps[0].size();
  Itype *d_in_map, *d_out_map;

  CUDA_CHECK(hipMalloc((void **)&d_in_map, 2 * nnz * sizeof(Itype)));
  d_out_map = d_in_map + nnz;

  CUDA_CHECK(hipMemcpy(d_in_map, in_maps[0].data(), sizeof(Itype) * nnz,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_out_map, out_maps[0].data(), sizeof(Itype) * nnz,
                        hipMemcpyHostToDevice));

  copy_in_out_map<Dtype, Itype>
      <<<GET_BLOCKS(nnz * nchannel), CUDA_NUM_THREADS, 0, stream>>>(
          nnz * nchannel, d_in_feat, d_out_feat, nchannel, d_in_map, d_out_map);

  hipFree(d_in_map);
}

template <typename Dtype, typename Itype>
void PruningBackwardKernelGPU(Dtype *d_grad_in_feat,
                              const Dtype *d_grad_out_feat, int nchannel,
                              const std::vector<std::vector<Itype>> &in_maps,
                              const std::vector<std::vector<Itype>> &out_maps,
                              hipStream_t stream) {
  int nnz = in_maps[0].size();
  Itype *d_in_map, *d_out_map;

  CUDA_CHECK(hipMalloc((void **)&d_in_map, 2 * nnz * sizeof(Itype)));
  d_out_map = d_in_map + nnz;

  CUDA_CHECK(hipMemcpy(d_in_map, in_maps[0].data(), sizeof(Itype) * nnz,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_out_map, out_maps[0].data(), sizeof(Itype) * nnz,
                        hipMemcpyHostToDevice));

  copy_in_out_map<Dtype, Itype>
      <<<GET_BLOCKS(nnz * nchannel), CUDA_NUM_THREADS, 0, stream>>>(
          nnz * nchannel, d_grad_out_feat, d_grad_in_feat, nchannel, d_out_map,
          d_in_map);

  hipFree(d_in_map);
}

template void PruningForwardKernelGPU<float, int32_t>(
    const float *d_in_feat, float *d_out_feat, int nchannel,
    const std::vector<std::vector<int32_t>> &in_maps,
    const std::vector<std::vector<int32_t>> &out_maps, hipStream_t stream);

template void PruningBackwardKernelGPU<float, int32_t>(
    float *d_grad_in_feat, const float *d_grad_out_feat, int nchannel,
    const std::vector<std::vector<int32_t>> &in_maps,
    const std::vector<std::vector<int32_t>> &out_maps, hipStream_t stream);
