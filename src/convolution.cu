#include "hip/hip_runtime.h"
/*  Copyright (c) Chris Choy (chrischoy@ai.stanford.edu).
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy of
 *  this software and associated documentation files (the "Software"), to deal in
 *  the Software without restriction, including without limitation the rights to
 *  use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies
 *  of the Software, and to permit persons to whom the Software is furnished to do
 *  so, subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in all
 *  copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 *  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 *  SOFTWARE.
 *
 *  Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 *  Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 *  of the code.
 */
#ifndef GPU_CONVOLUTION
#define GPU_CONVOLUTION

#include <iostream>

// Use the torch for GPU memory management. Thrust resize gives segfulat during
// debugging -g #include <torch/extension.h>

#include "convolution.cuh"

// Given a row-major matrix, use the mapping to extract a row major order matrix
template <typename Dtype, typename Itype>
__global__ void copy_mapped_input(const int n, const int nchannel,
                                  const Dtype *in_feat, Dtype *out_feat,
                                  const Itype *map) {
  CUDA_KERNEL_LOOP(index, n) {
    const int row = index / nchannel;
    const int col = index % nchannel;
    out_feat[index] = in_feat[map[row] * nchannel + col];
  }
}

template <typename Dtype, typename Itype>
__global__ void add_mapped_output_tr(const int n, const Dtype *in_feat,
                                     const int in_nchannel, Dtype *out_feat,
                                     const int out_nchannel, const Itype *map) {
  CUDA_KERNEL_LOOP(index, n) {
    const int row = index % in_nchannel;
    const int col = index / in_nchannel;
    atomicAdd(&out_feat[map[row] * out_nchannel + col], in_feat[index]);
  }
}

// Given each output, get an input feature for each corresponding kernel weight
// and add the output in place
template <typename Dtype, typename Itype>
__global__ void inplace_convolution(const int n, const Dtype *in_feat,
                                    const int in_nchannel, Dtype *out_feat,
                                    const int out_nchannel, const Dtype *kernel,
                                    const Itype *in_map, const Itype *out_map) {
  // n = out_nchannel * out_nrows
  // The kernel computes one output scalar for each output index and each output
  // channel.
  CUDA_KERNEL_LOOP(index, n) {
    const int out_ch = index % out_nchannel;
    const int out_row = index / out_nchannel;
    // Pytorch tensors in C-ordering with in_nchannels x out_nchannels
    Dtype tmp = 0.0;
    const Dtype *curr_kernel = kernel + out_ch;
    const Dtype *curr_in_feat = in_feat + in_map[out_row] * in_nchannel;
    for (int in_ch = 0; in_ch < in_nchannel; in_ch++) {
      tmp += (*curr_kernel) * (*curr_in_feat);
      curr_kernel += out_nchannel;
      curr_in_feat += 1;
    }
    // Done independently, no need for atomicAdd
    out_feat[out_map[out_row] * out_nchannel + out_ch] += tmp;
  }
}

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <typename Dtype, typename Itype, int BLOCK_SIZE>
__global__ void matmul(const Dtype *A, const int wA, const int hA,
                       const Dtype *B, const int wB, const int hB, Dtype *C,
                       const Itype *in_map, const Itype *out_map) {
  // Use in_feat as A and kernel as B

  // Block index
  const int bx = blockIdx.x;
  const int by = blockIdx.y;

  // Thread index
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;

  // Coordinate. x is for rows, y is for columns.
  const int x = BLOCK_SIZE * bx + tx;
  const int y = BLOCK_SIZE * by + ty;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  Dtype Csub = 0;

  const Itype in_row = y < hA ? in_map[y] : 0;
  const Itype out_row = y < hA ? out_map[y] : 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int s = 0; s < wA; s += BLOCK_SIZE) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ Dtype As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ Dtype Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = ((s + tx) < wA && y < hA) ? A[wA * in_row + s + tx] : 0;
    Bs[ty][tx] = ((s + ty) < hB && x < wB) ? B[wB * (s + ty) + x] : 0;

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty][k] * Bs[k][tx];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  if (y < hA && x < wB)
    C[wB * out_row + x] += Csub;
  // TODO: atomicAdd(&C[wB * out_row + x], Csub); // For conv transpose, it
  // might fail due to overlapping outputs
}

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B^T, E = D^T * A
 * wA is A's width and wB is B's width
 *
 *                +---+
 *                |B^T|
 *            +-------+
 *            |   |   |
 *            | A | C |
 *            |   |   |
 *            |   |   |
 * +------------------+
 * |    D^T   | E |
 * +----------+---+
 *
 */
template <typename Dtype, typename Itype, int BLOCK_SIZE>
__global__ void matmul2(const Dtype *A, const int wA, const int hA,
                        const Dtype *B, const int wB, const int hB,
                        const Dtype *D, const int wD, const int hD, Dtype *C,
                        Dtype *E, const Itype *in_map, const Itype *out_map) {
  // Use grad_out_feat as A, transposed kernel weight as B, and in_feat as D

  // Block index
  const int bx = blockIdx.x;
  const int by = blockIdx.y;

  // Thread index
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;

  // Coordinate. x is for rows, y is for columns.
  const int x = BLOCK_SIZE * bx + tx;
  const int y = BLOCK_SIZE * by + ty;

  const Itype in_row = y < hA ? in_map[y] : 0;
  const Itype out_row = y < hA ? out_map[y] : 0;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  Dtype Csub = 0;
  Dtype Esub = 0;

  // Declaration of the shared memory array As used to
  // store the sub-matrix of A
  __shared__ Dtype As[BLOCK_SIZE][BLOCK_SIZE];

  // Declaration of the shared memory array Bs used to
  // store the sub-matrix of B
  __shared__ Dtype BTs[BLOCK_SIZE][BLOCK_SIZE];

  // Declaration of the shared memory array Ds used to
  // store the sub-matrix of D
  __shared__ Dtype DTs[BLOCK_SIZE][BLOCK_SIZE];

  // For Ds = D^T[...:..., ...:...], use the transposed grid dimension for A
  DTs[ty][tx] = (x < wD && y < hD) ? D[wD * in_row + x] : 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int s = 0; s < wA; s += BLOCK_SIZE) {
    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = ((s + tx) < wA && y < hA) ? A[wA * out_row + s + tx] : 0;

    // Transposed kernel
    BTs[ty][tx] = ((s + ty) < wB && x < hB) ? B[wB * x + s + ty] : 0;

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty][k] * BTs[k][tx];
    }

    // For Esub, reset to 0
    Esub = 0;
#pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Esub += DTs[k][ty] * As[k][tx];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();

    // For the E matrix which requires accmulation of multiple blocks, use
    // atomic addition. This can be replaced with a more sophisticaed reduction
    // algorithm.
    if ((bx * BLOCK_SIZE + ty) < wD && (s + tx) < wA)
      atomicAdd(&E[wA * (bx * BLOCK_SIZE + ty) + (s + tx)], Esub);
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  if (y < hA && x < hB)
    atomicAdd(&C[hB * in_row + x], Csub);
}

template <typename Dtype, typename Itype>
void ConvolutionForwardKernelGPU(
    const Dtype *d_in_feat, int in_nchannel, Dtype *d_out_feat,
    int out_nchannel, const Dtype *d_kernel,
    const std::vector<std::vector<Itype>> &in_maps,
    const std::vector<std::vector<Itype>> &out_maps, int out_nrows,
    hipblasHandle_t cuhandle, hipStream_t stream) {
  // For the in out buffer, use the pre allocated GPU memory space as thrust
  // resize gives segfault. Also initializing it with torch allows us to
  // allocate memory faster and efficiently.
  int kernel_volume, n_active_in_volume, num_kernels;
  Itype *d_in_map, *d_out_map;
  // Copy the in_map, out_map to GPU
  kernel_volume = in_maps.size();

  // Find the max_n_active fot memory allocation
  int max_n_active = -1;
  for (int k = 0; k < kernel_volume; k++)
    if (max_n_active < (int)(in_maps[k].size()))
      max_n_active = (int)(in_maps[k].size());

  // Create a large chunk of memory
  CUDA_CHECK(
      hipMalloc((void **)&d_in_map, (2 * max_n_active) * sizeof(Itype)));
  d_out_map = d_in_map + max_n_active;

  // Iterate through each spatial kernel and get indices for in_map and out_map
  for (int k = 0; k < kernel_volume; k++) {
    n_active_in_volume = in_maps[k].size();
    if (n_active_in_volume == 0)
      continue;

    // Copy (*p_in_maps)[k] to GPU
    CUDA_CHECK(hipMemcpy(d_in_map, in_maps[k].data(),
                          sizeof(Itype) * n_active_in_volume,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_out_map, out_maps[k].data(),
                          sizeof(Itype) * n_active_in_volume,
                          hipMemcpyHostToDevice));

    if (n_active_in_volume / SHARED_BLOCK_SIZE < 65536) {
      dim3 threads(SHARED_BLOCK_SIZE, SHARED_BLOCK_SIZE);
      dim3 grid((out_nchannel + threads.x - 1) / threads.x,
                (n_active_in_volume + threads.y - 1) / threads.y);
      matmul<Dtype, Itype, SHARED_BLOCK_SIZE><<<grid, threads, 0, stream>>>(
          d_in_feat, in_nchannel, n_active_in_volume,
          &d_kernel[k * in_nchannel * out_nchannel], out_nchannel, in_nchannel,
          d_out_feat, d_in_map, d_out_map);
    } else {
      num_kernels = out_nchannel * n_active_in_volume;
      inplace_convolution<Dtype, Itype>
          <<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS, 0, stream>>>(
              num_kernels, d_in_feat, in_nchannel, d_out_feat, out_nchannel,
              &d_kernel[k * in_nchannel * out_nchannel], d_in_map, d_out_map);
    }
  }

  hipFree(d_in_map);
}

template void ConvolutionForwardKernelGPU<float, int32_t>(
    const float *d_in_feat, int in_nchannel, float *d_out_feat,
    int out_nchannel, const float *d_kernel,
    const std::vector<std::vector<int32_t>> &in_map,
    const std::vector<std::vector<int32_t>> &out_map, int out_nrows,
    hipblasHandle_t cuhandle, hipStream_t stream);

template void ConvolutionForwardKernelGPU<double, int32_t>(
    const double *d_in_feat, int in_nchannel, double *d_out_feat,
    int out_nchannel, const double *d_kernel,
    const std::vector<std::vector<int32_t>> &in_map,
    const std::vector<std::vector<int32_t>> &out_map, int out_nrows,
    hipblasHandle_t cuhandle, hipStream_t stream);

template <typename Dtype, typename Itype>
void ConvolutionBackwardKernelGPU(
    const Dtype *d_in_feat, Dtype *d_grad_in_feat, int in_nchannel,
    const Dtype *d_grad_out_feat, int out_nchannel, const Dtype *d_kernel,
    Dtype *d_grad_kernel, const std::vector<std::vector<Itype>> &in_maps,
    const std::vector<std::vector<Itype>> &out_maps, int out_nrows,
    hipblasHandle_t cuhandle, hipStream_t stream) {
  int kernel_volume, n_active_in_volume, num_kernels;
  Itype *d_in_map, *d_out_map;
  Dtype *d_in_buffer, *d_out_buffer;

  kernel_volume = in_maps.size();
  // Find the max_n_active fot memory allocation
  int max_n_active = -1;
  for (int k = 0; k < kernel_volume; k++)
    if (max_n_active < (int)(in_maps[k].size()))
      max_n_active = (int)(in_maps[k].size());

  CUDA_CHECK(hipMalloc((void **)&d_in_map, 2 * max_n_active * sizeof(Itype)));
  d_out_map = d_in_map + max_n_active;

  // Use the old kernel when grid-y dim exceeds the limit.
  if (max_n_active / SHARED_BLOCK_SIZE >= 65536) {
    CUDA_CHECK(hipMalloc((void **)&d_in_buffer, (in_nchannel + out_nchannel) *
                                                     max_n_active *
                                                     sizeof(Dtype)));
    d_out_buffer = d_in_buffer + in_nchannel * max_n_active;
  }

  for (int k = 0; k < kernel_volume; k++) {
    n_active_in_volume = in_maps[k].size();
    if (n_active_in_volume == 0)
      continue;

    // Copy (*p_in_maps)[k] to GPU
    CUDA_CHECK(hipMemcpy(d_in_map, in_maps[k].data(),
                          sizeof(Itype) * n_active_in_volume,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_out_map, out_maps[k].data(),
                          sizeof(Itype) * n_active_in_volume,
                          hipMemcpyHostToDevice));

    if (n_active_in_volume / SHARED_BLOCK_SIZE < 65536) {
      dim3 threads(SHARED_BLOCK_SIZE, SHARED_BLOCK_SIZE);
      dim3 grid((in_nchannel + threads.x - 1) / threads.x,
                (n_active_in_volume + threads.y - 1) / threads.y);

      matmul2<Dtype, Itype, SHARED_BLOCK_SIZE><<<grid, threads, 0, stream>>>(
          d_grad_out_feat, out_nchannel, n_active_in_volume, // A
          &d_kernel[k * in_nchannel * out_nchannel], out_nchannel,
          in_nchannel,                                    // B
          d_in_feat, in_nchannel, n_active_in_volume,     // D
          d_grad_in_feat,                                 // C
          &d_grad_kernel[k * in_nchannel * out_nchannel], // E
          d_in_map, d_out_map);

    } else {
      // Copy (*p_in_maps)[k] to GPU
      num_kernels = out_nchannel * n_active_in_volume;

      // Copy gradients to the buffer
      copy_mapped_input<Dtype, Itype>
          <<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS, 0, stream>>>(
              num_kernels, out_nchannel, d_grad_out_feat, d_out_buffer,
              d_out_map);

      gpu_gemm<Dtype>(cuhandle, CblasNoTrans, CblasTrans,
                      in_nchannel,                               // M
                      n_active_in_volume,                        // N
                      out_nchannel,                              // K
                      (Dtype)1.,                                 // alpha
                      &d_kernel[k * in_nchannel * out_nchannel], // A
                      d_out_buffer,                              // B
                      (Dtype)0.,                                 // beta
                      d_in_buffer);                              // C

      // Accumulate gradients back to the input grad feat
      // Put it back to the correct index
      num_kernels = in_nchannel * n_active_in_volume;
      add_mapped_output_tr<Dtype>
          <<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS, 0, stream>>>(
              num_kernels,
              d_in_buffer,                 // In
              n_active_in_volume,          // In channel
              d_grad_in_feat, in_nchannel, // Out
              d_in_map);                   // Out channel

      // Compute gradient for kernel
      // Copy features to the buffer
      copy_mapped_input<Dtype, Itype>
          <<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS, 0, stream>>>(
              num_kernels, in_nchannel, d_in_feat, d_in_buffer, d_in_map);

      gpu_gemm<Dtype>(cuhandle, CblasTrans, CblasNoTrans,
                      in_nchannel,                                   // M
                      out_nchannel,                                  // N
                      n_active_in_volume,                            // K
                      1,                                             // alpha
                      d_in_buffer,                                   // A
                      d_out_buffer,                                  // B
                      1,                                             // beta
                      &d_grad_kernel[k * in_nchannel * out_nchannel] // C
      );
    }
  }
  hipFree(d_in_map);

  // Free the mem allocated when grid-y dim exceeds the limit.
  if (max_n_active / SHARED_BLOCK_SIZE >= 65536)
    hipFree(d_in_buffer);
}

template void ConvolutionBackwardKernelGPU<float, int32_t>(
    const float *d_in_feat, float *d_grad_in_feat, int in_nchannel,
    const float *d_grad_out_feat, int out_nchannel, const float *d_kernel,
    float *p_grad_kernel, const std::vector<std::vector<int32_t>> &in_map,
    const std::vector<std::vector<int32_t>> &out_map, int out_nrows,
    hipblasHandle_t cuhandle, hipStream_t stream);

template void ConvolutionBackwardKernelGPU<double, int32_t>(
    const double *d_in_feat, double *d_grad_in_feat, int in_nchannel,
    const double *d_grad_out_feat, int out_nchannel, const double *d_kernel,
    double *p_grad_kernel, const std::vector<std::vector<int32_t>> &in_map,
    const std::vector<std::vector<int32_t>> &out_map, int out_nrows,
    hipblasHandle_t cuhandle, hipStream_t stream);
#endif
