#include <cstdio>
#include <iomanip>
#include <iostream>

#include "gpu.cuh"

template <typename Dtype> void print(const thrust::device_vector<Dtype> &v) {
  for (size_t i = 0; i < v.size(); i++)
    std::cout << " " << std::fixed << std::setprecision(3) << v[i];
  std::cout << "\n";
}

template void print(const thrust::device_vector<float> &v);
template void print(const thrust::device_vector<int32_t> &v);

template <typename Dtype1, typename Dtype2>
void print(const thrust::device_vector<Dtype1> &v1,
           const thrust::device_vector<Dtype2> &v2) {
  for (size_t i = 0; i < v1.size(); i++)
    std::cout << " (" << v1[i] << "," << std::setw(2) << v2[i] << ")";
  std::cout << "\n";
}

template void print(const thrust::device_vector<int32_t> &v1,
                    const thrust::device_vector<int32_t> &v2);

const char* cublasGetErrorString(hipblasStatus_t error) {
  switch (error) {
  case HIPBLAS_STATUS_SUCCESS:
    return "HIPBLAS_STATUS_SUCCESS";
  case HIPBLAS_STATUS_NOT_INITIALIZED:
    return "HIPBLAS_STATUS_NOT_INITIALIZED";
  case HIPBLAS_STATUS_ALLOC_FAILED:
    return "HIPBLAS_STATUS_ALLOC_FAILED";
  case HIPBLAS_STATUS_INVALID_VALUE:
    return "HIPBLAS_STATUS_INVALID_VALUE";
  case HIPBLAS_STATUS_ARCH_MISMATCH:
    return "HIPBLAS_STATUS_ARCH_MISMATCH";
  case HIPBLAS_STATUS_MAPPING_ERROR:
    return "HIPBLAS_STATUS_MAPPING_ERROR";
  case HIPBLAS_STATUS_EXECUTION_FAILED:
    return "HIPBLAS_STATUS_EXECUTION_FAILED";
  case HIPBLAS_STATUS_INTERNAL_ERROR:
    return "HIPBLAS_STATUS_INTERNAL_ERROR";
#if CUDA_VERSION >= 6000
  case HIPBLAS_STATUS_NOT_SUPPORTED:
    return "HIPBLAS_STATUS_NOT_SUPPORTED";
#endif
#if CUDA_VERSION >= 6050
  case HIPBLAS_STATUS_UNKNOWN:
    return "HIPBLAS_STATUS_UNKNOWN";
#endif
  }
  return "Unknown cublas status";
}

const char* hipsparseGetErrorString(hipsparseStatus_t error) {
  // Read more at: http://docs.nvidia.com/cuda/cusparse/index.html#ixzz3f79JxRar
  switch (error) {
  case HIPSPARSE_STATUS_SUCCESS:
    return "The operation completed successfully.";
  case HIPSPARSE_STATUS_NOT_INITIALIZED:
    return "HIPSPARSE_STATUS_NOT_INITIALIZED";

  case HIPSPARSE_STATUS_ALLOC_FAILED:
    return "HIPSPARSE_STATUS_ALLOC_FAILED";

  case HIPSPARSE_STATUS_INVALID_VALUE:
    return "HIPSPARSE_STATUS_INVALID_VALUE";

  case HIPSPARSE_STATUS_ARCH_MISMATCH:
    return "HIPSPARSE_STATUS_ARCH_MISMATCH";

  case HIPSPARSE_STATUS_MAPPING_ERROR:
    return "HIPSPARSE_STATUS_MAPPING_ERROR";

  case HIPSPARSE_STATUS_EXECUTION_FAILED:
    return "HIPSPARSE_STATUS_EXECUTION_FAILED";

  case HIPSPARSE_STATUS_INTERNAL_ERROR:
    return "HIPSPARSE_STATUS_INTERNAL_ERROR";

  case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
    return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
  }

  return "<unknown>";
}
