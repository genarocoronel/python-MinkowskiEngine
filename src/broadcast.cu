#include "hip/hip_runtime.h"
#ifndef GPU_BROADCAST
#define GPU_BROADCAST

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "broadcast.cuh"
#include "math_functions.hpp"

template <class T> struct IsIntType { static const bool value = false; };

template <> struct IsIntType<int> { static const bool value = true; };

template <typename Dtype, typename Itype>
__global__ void
channelwise_addition(const int n, const int nchannel, const Dtype *d_glob_feat,
                     const Itype *d_sorted_map, Dtype *d_out_feat) {
  int row, ch_index;
  CUDA_KERNEL_LOOP(index, n) {
    ch_index = index % nchannel;
    row = d_sorted_map[index / nchannel];
    d_out_feat[index] += d_glob_feat[row * nchannel + ch_index];
  }
}

template <typename Dtype, typename Itype>
__global__ void channelwise_multiplication(const int n, const int nchannel,
                                           const Dtype *d_glob_feat,
                                           const Itype *d_sorted_out_map,
                                           Dtype *d_out_feat) {
  int row, ch_index;
  CUDA_KERNEL_LOOP(index, n) {
    ch_index = index % nchannel;
    row = d_sorted_out_map[index / nchannel];
    d_out_feat[index] *= d_glob_feat[row * nchannel + ch_index];
  }
}

template <typename Dtype, typename Itype>
__global__ void
channelwise_division(const int n, const int nchannel, const Dtype *d_glob_feat,
                     const Itype *d_sorted_out_map, Dtype *d_out_feat) {
  int row, ch_index;
  CUDA_KERNEL_LOOP(index, n) {
    ch_index = index % nchannel;
    row = d_sorted_out_map[index / nchannel];
    d_out_feat[index] /= d_glob_feat[row * nchannel + ch_index];
  }
}

template <typename Dtype>
__global__ void fill(const int n, Dtype *in_feat, Dtype val) {
  CUDA_KERNEL_LOOP(index, n) { in_feat[index] = val; }
}

template <typename Dtype, typename Itype>
void BroadcastForwardKernelGPU(
    const Dtype *d_in_feat, int in_nrows, const Dtype *d_in_feat_global,
    int in_nrows_global, Dtype *d_out_feat, int nchannel, int op,
    const std::vector<std::vector<Itype>> &sorted_in_map,
    const std::vector<std::vector<Itype>> &sorted_out_map,
    hipsparseHandle_t cushandle, hipStream_t stream) {
  thrust::device_vector<Itype> d_sorted_out_map;
  // Copy all in_feat to out_feat
  CUDA_CHECK(hipMemcpy(d_out_feat, d_in_feat,
                        sizeof(Dtype) * nchannel * in_nrows,
                        hipMemcpyDeviceToDevice));

  if (sorted_in_map.size() != 1)
    throw std::invalid_argument("InOut map must have one kernel for Broadcast");

  if (sorted_in_map[0].size() != in_nrows) {
    std::cout << "sorted_in_map[0].size(): " << sorted_in_map[0].size()
              << ", in_nrows: " << in_nrows << std::endl;
    throw std::invalid_argument("Invalid in_map");
  }

  THRUST_CHECK(d_sorted_out_map = sorted_out_map[0]);

  // To speed up, put switch outside for loops
  switch (op) {
  case 0: // +
    channelwise_addition<Dtype, Itype>
        <<<GET_BLOCKS(in_nrows * nchannel), CUDA_NUM_THREADS, 0, stream>>>(
            nchannel * in_nrows, nchannel, d_in_feat_global,
            thrust::raw_pointer_cast(d_sorted_out_map.data()), d_out_feat);
    break;
  case 1: // *
    channelwise_multiplication<Dtype, Itype>
        <<<GET_BLOCKS(in_nrows * nchannel), CUDA_NUM_THREADS, 0, stream>>>(
            nchannel * in_nrows, nchannel, d_in_feat_global,
            thrust::raw_pointer_cast(d_sorted_out_map.data()), d_out_feat);
    break;
  default:
    throw std::invalid_argument(Formatter() << "Operation not supported: "
                                            << std::to_string(op));
  }
}

template void BroadcastForwardKernelGPU<float, int32_t>(
    const float *d_in_feat, int in_nrows, const float *d_in_feat_global,
    int in_nrows_global, float *d_out_feat, int nchannel, int op,
    const std::vector<std::vector<int32_t>> &sorted_in_map,
    const std::vector<std::vector<int32_t>> &sorted_out_map,
    hipsparseHandle_t cuhandle, hipStream_t stream);

template <typename Dtype, typename Itype>
void BroadcastBackwardKernelGPU(
    const Dtype *d_in_feat, Dtype *d_grad_in_feat, int in_nrows,
    const Dtype *d_in_feat_global, Dtype *d_grad_in_feat_global,
    int in_nrows_global, const Dtype *d_grad_out_feat, int nchannel, int op,
    const std::vector<std::vector<Itype>> &sorted_in_maps,
    const std::vector<std::vector<Itype>> &sorted_out_maps,
    hipsparseHandle_t cushandle, hipStream_t stream) {
  Itype *d_sorted_in_map, *d_sorted_out_map, *d_csr_row;
  Dtype *d_dtype, *d_csr_val, *d_tmp_grad_in_feat_global, *d_tmp_grad_in_feat;
  hipsparseMatDescr_t descr = 0;
  const Dtype alpha = 1;
  const Dtype beta = 0;
  int nnz = in_nrows;

  if (!IsIntType<Itype>::value)
    throw std::invalid_argument("Not implemented"); // Due to hipsparseXcoo2csr

  if (sorted_in_maps.size() != 1)
    throw std::invalid_argument("InOut map must have one kernel for Broadcast");

  if (sorted_in_maps[0].size() != in_nrows)
    throw std::invalid_argument("Invalid in_map");

  // Malloc d_sorted_in_map, d_sorted_out_map, d_csr_row
  // THRUST_CHECK(d_csr_row.resize(in_nrows_global + 1)); // CSR returns n_row +
  // 1
  CUDA_CHECK(hipMalloc((void **)&d_sorted_in_map,
                        (sorted_in_maps[0].size() + sorted_out_maps[0].size() +
                         in_nrows_global + 1) *
                            sizeof(Itype)));
  d_sorted_out_map = d_sorted_in_map + sorted_in_maps[0].size();
  d_csr_row = d_sorted_out_map + sorted_out_maps[0].size();

  // d_tmp_grad_in_feat, d_tmp_grad_in_feat_global
  // THRUST_CHECK(d_tmp_grad_in_feat.resize(in_nrows * nchannel));
  // THRUST_CHECK(d_csr_val.resize(nnz));
  CUDA_CHECK(hipMalloc((void **)&d_dtype,
                        (nnz + (in_nrows + in_nrows_global) * nchannel) *
                            sizeof(Dtype)));
  d_tmp_grad_in_feat_global = d_dtype;
  d_tmp_grad_in_feat = d_tmp_grad_in_feat_global + in_nrows_global * nchannel;
  d_csr_val = d_tmp_grad_in_feat + in_nrows * nchannel;

  // COO cols
  // THRUST_CHECK(d_sorted_in_map = sorted_in_map[0]);    // COO cols
  hipMemcpy(d_sorted_in_map, sorted_in_maps[0].data(),
             sizeof(Itype) * sorted_in_maps[0].size(), hipMemcpyHostToDevice);
  // COO rows
  // THRUST_CHECK(d_sorted_out_map = sorted_out_map[0]);  // COO rows
  hipMemcpy(d_sorted_out_map, sorted_out_maps[0].data(),
             sizeof(Itype) * sorted_out_maps[0].size(), hipMemcpyHostToDevice);

  // thrust::fill(d_csr_val.begin(), d_csr_val.end(), 1);
  fill<Dtype><<<GET_BLOCKS(in_nrows), CUDA_NUM_THREADS, 0, stream>>>(
      nnz, d_csr_val, (Dtype)1.);

  CUSPARSE_CHECK(hipsparseCreateMatDescr(&descr));
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  // Sort COO first
  sort_coo_gpu(cushandle, in_nrows_global, in_nrows, nnz, d_sorted_out_map,
               d_sorted_in_map);
  // For CRS, sort row and col inds by row major.
  CUSPARSE_CHECK(hipsparseXcoo2csr(cushandle, d_sorted_out_map, nnz,
                                  in_nrows_global, d_csr_row,
                                  HIPSPARSE_INDEX_BASE_ZERO));

  // To speed up, put switch outside for loops
  switch (op) {
  case 0: // +
    // For grad_in_feat, copy all grad_out_feat to grad_in_feat
    CUDA_CHECK(hipMemcpy(d_grad_in_feat, d_grad_out_feat,
                          sizeof(Dtype) * nchannel * in_nrows,
                          hipMemcpyDeviceToDevice));
    // For grad_in_feat_glob, add all grad_out_feat
    CUSPARSE_CHECK(
        cusparse_csrmm<Dtype>(cushandle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE, // op(A)
                              HIPSPARSE_OPERATION_TRANSPOSE,     // op(B)
                              in_nrows_global,                  // M
                              nchannel,                         // N
                              in_nrows,                         // K
                              nnz, &alpha, descr,
                              d_csr_val,       // val
                              d_csr_row,       // row
                              d_sorted_in_map, // col
                              d_grad_out_feat, // B
                              nchannel,        // ldb
                              &beta,
                              d_tmp_grad_in_feat_global, // C
                              in_nrows_global            // ldc
                              ));

    col2row_major<Dtype>(in_nrows_global, nchannel, d_tmp_grad_in_feat_global,
                         d_grad_in_feat_global, stream);
    break;
  case 1: // *
    // First, for grad_in_feat
    // Copy in_feat_global to tmp, then multiply the tmp with grad_out_feat
    row2col_major<Dtype>(in_nrows_global, nchannel, d_in_feat_global,
                         d_tmp_grad_in_feat_global, stream);
    CUSPARSE_CHECK(
        cusparse_csrmm<Dtype>(cushandle,
                              HIPSPARSE_OPERATION_TRANSPOSE,     // op(A)
                              HIPSPARSE_OPERATION_NON_TRANSPOSE, // op(B)
                              in_nrows_global,                  // M
                              nchannel,                         // N
                              in_nrows,                         // K
                              nnz, &alpha, descr,
                              d_csr_val,                 // val
                              d_csr_row,                 // row
                              d_sorted_in_map,           // col
                              d_tmp_grad_in_feat_global, // B
                              in_nrows_global,           // ldb
                              &beta,
                              d_tmp_grad_in_feat, // C
                              in_nrows            // ldc
                              ));
    col2row_major<Dtype>(in_nrows, nchannel, d_tmp_grad_in_feat, d_grad_in_feat,
                         stream);
    gpu_multiplication<Dtype>(nchannel * in_nrows, d_grad_out_feat,
                              d_grad_in_feat, d_grad_in_feat, stream);

    // Second, for grad_in_feat_global, copy in_feat to tmp,
    CUDA_CHECK(hipMemcpy(d_tmp_grad_in_feat, d_grad_out_feat,
                          sizeof(Dtype) * nchannel * in_nrows,
                          hipMemcpyDeviceToDevice));
    gpu_multiplication<Dtype>(nchannel * in_nrows, d_in_feat,
                              d_tmp_grad_in_feat, d_tmp_grad_in_feat, stream);
    CUSPARSE_CHECK(
        cusparse_csrmm<Dtype>(cushandle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE, // op(A)
                              HIPSPARSE_OPERATION_TRANSPOSE,     // op(B)
                              in_nrows_global,                  // M
                              nchannel,                         // N
                              in_nrows,                         // K
                              nnz, &alpha, descr,
                              d_csr_val,          // val
                              d_csr_row,          // row
                              d_sorted_in_map,    // col
                              d_tmp_grad_in_feat, // B
                              nchannel,           // ldb
                              &beta,
                              d_tmp_grad_in_feat_global, // C
                              in_nrows_global            // ldc
                              ));
    col2row_major<Dtype>(in_nrows_global, nchannel, d_tmp_grad_in_feat_global,
                         d_grad_in_feat_global, stream);
    break;
  default:
    throw std::invalid_argument(Formatter() << "Operation not supported: "
                                            << std::to_string(op));
  }

  CUSPARSE_CHECK(hipsparseDestroyMatDescr(descr));

  hipFree(d_sorted_in_map);
  hipFree(d_dtype);
}

template void BroadcastBackwardKernelGPU<float, int32_t>(
    const float *d_in_feat, float *d_grad_in_feat, int in_nrows,
    const float *d_in_feat_global, float *d_grad_in_feat_global,
    int in_nrows_global, const float *d_grad_out_feat, int nchannel, int op,
    const std::vector<std::vector<int32_t>> &sorted_in_map,
    const std::vector<std::vector<int32_t>> &sorted_out_map,
    hipsparseHandle_t cushandle, hipStream_t stream);

#endif
